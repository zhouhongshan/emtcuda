#include <stdio.h>
#include<hip/hip_runtime.h>
#include <time.h>


// CUDA runtime


// Helper functions and utilities to work with CUDA


#define N 256
//#define M 256


//__global__�����ĺ��������߱�������δ��뽻��CPU���ã���GPUִ��
__global__ void matrix_mult(float *dev_a, float* dev_b, float* dev_c, int Width)
{
	int Row = blockIdx.y*blockDim.y+threadIdx.y;
	int Col = blockIdx.x*blockDim.x+threadIdx.x;
	if ((Row < Width) && (Col < Width)) {
	float Pvalue = 0;
	for (int k = 0; k < Width; k++)
	{
		Pvalue += dev_a[Row*Width + k] * dev_b[k*Width+Col];
	}
	dev_c[Row*Width + Col] = Pvalue;

}
}

int main(void)
{
	//���������ڴ棬�����г�ʼ��
	//clock_t start = clock();
	float host_a[N][N];
	float host_b[N][N];
	float host_c[N][N];
	for (int i = 0; i<N; i++)
	    for (int j = 0; j<N; j++)
	     host_a[i][j] = 1.0f;
	for (int i = 0; i<N; i++)
		for (int j = 0; j<N; j++)
		host_b[i][j] = 0.01f;
		/*
	for (int i = 0; i < N; i++)
	for (int j = 0; j < N; j++)
		printf("%d\n", host_a[i][j]);
	*/

	//����cudaError��Ĭ��ΪcudaSuccess(0)
	hipError_t err = hipSuccess;

	//����GPU�洢�ռ�
	float *dev_a, *dev_b, *dev_c;
	err = hipMalloc((void **)&dev_a, sizeof(float)* N*N);
	err = hipMalloc((void **)&dev_b, sizeof(float)* N*N);
	err = hipMalloc((void **)&dev_c, sizeof(float)* N*N);
	if (err != hipSuccess)
	{
		printf("the hipMalloc on GPU is failed");
		return 1;
	}
	printf("SUCCESS");
	//��Ҫ���������ʹ��cudaMemcpy���͵�GPU
	hipMemcpy(dev_a, host_a, sizeof(float)* N*N, hipMemcpyHostToDevice);
	hipMemcpy(dev_b, host_b, sizeof(float)* N*N, hipMemcpyHostToDevice);
	//cudaMemcpy(dev_a, host_a, sizeof(host_a), cudaMemcpyHostToDevice);
	//cudaMemcpy(dev_b, host_b, sizeof(host_b), cudaMemcpyHostToDevice);
	
	//���ú˺�����GPU��ִ�С����ݽ��٣�֮ʹ��һ��Block������1024���߳�
    #define BLOCK_WIDTH 32
	int NumBlocks = N / BLOCK_WIDTH;
	//int NumBlocks2 = M / BLOCK_WIDTH;
	if ( N%BLOCK_WIDTH ) NumBlocks++;
	//if (M%BLOCK_WIDTH) NumBlocks2++;
		dim3 dimGrid(NumBlocks, NumBlocks);
	    dim3 dimBlock(BLOCK_WIDTH, BLOCK_WIDTH);
		clock_t start = clock();
	matrix_mult<<< dimGrid, dimBlock >>>(dev_a, dev_b, dev_c, N);
	hipMemcpy(&host_c, dev_c, sizeof(host_c), hipMemcpyDeviceToHost);

	clock_t end = clock();
	float time = (float)(end - start) / CLOCKS_PER_SEC;
	printf("%f seconds\n", time);

	//for (int i = 0; i < N; i++)
	//for (int j = 0; j < N;j++)
		//printf("%f\n", host_c[i][j]);

	hipFree(dev_a);//�ͷ�GPU�ڴ�
	hipFree(dev_b);//�ͷ�GPU�ڴ�
	hipFree(dev_c);//�ͷ�GPU�ڴ�

	

	return 0;
}

